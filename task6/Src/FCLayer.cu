#include <iostream>
#include <cstdio>

#include <hip/hip_runtime.h>

#include "../Inc/FCLayer.cuh"
#include "../Inc/Functions.cuh"
#include "../Inc/Errors.cuh"

Linear::Linear(hipblasHandle_t handle, const char* pathToWeights, int in, int out) : 
        cublasHandle(handle), sizeY(in), sizeX(out)
{
    // Allocate memory
    float* tempBufferForWeights;
    GET_CUDA_STATUS(hipHostMalloc(&tempBufferForWeights, sizeof(float) * in * out));
    GET_CUDA_STATUS(hipMalloc(&this->weights, sizeof(float) * in * out));
    GET_CUDA_STATUS(hipMalloc(&this->output, sizeof(float) * out));

    // Here we will write weights from 'pathToWeights' file
    FILE* fin = std::fopen(pathToWeights, "rb");
    if (!fin)
    {
        std::cout << "There's no such file: " << pathToWeights << std::endl;
        std::exit(-1);
    }

    std::fread(tempBufferForWeights, sizeof(float), in * out, fin);

    GET_CUDA_STATUS(hipMemcpy(
        (void*)this->weights, 
        (void*)tempBufferForWeights, 
        sizeof(float) * in * out,
        hipMemcpyHostToDevice));

    // Delete temp buffer 
    GET_CUDA_STATUS(hipHostFree(tempBufferForWeights));
    std::fclose(fin);
}

Linear::~Linear()
{
    if (this->output)   GET_CUDA_STATUS(hipFree(this->output));
    if (this->weights)  GET_CUDA_STATUS(hipFree(this->weights));
}

void Linear::forward(float* input, float** output)
{
    const float alpha = 1.0, beta = 0.0;

    GET_CUBLAS_STATUS(hipblasSgemv(
        this->cublasHandle,
        HIPBLAS_OP_T,
        this->sizeY, 
        this->sizeX,
        &alpha,
        this->weights,
        this->sizeY,
        input, 
        1,
        &beta,
        this->output,
        1));

    *output = this->output;
}

 int Linear::getInputSize()
 {
    return this->sizeY;
 }

 int Linear::getOutputSize()
 {
    return this->sizeX;
 }

 

