#include <iostream>
#include <vector>
#include <memory>

#include "../Inc/FCLayer.cuh"
#include "../Inc/Functions.cuh"

// Class representing a neural network
class Network
{
public:
    // Constructor that initializes the network 
    Network(hipblasHandle_t handle, std::vector<LinearArguments>& args)
    {
        // Create three fully connected layers with specified input and output sizes
        this->fc1 = std::make_unique<Linear>(handle, args[0]);
        this->fc2 = std::make_unique<Linear>(handle, args[1]);
        this->fc3 = std::make_unique<Linear>(handle, args[2]);
    }

    ~Network() = default;

    void forward(float* input, float* output)
    {
        float* layerOutputPtr = nullptr;
        
        // Perform forward pass through the network layers
        this->fc1->forward(input, &layerOutputPtr);
        sigmoid(layerOutputPtr, this->fc1->getOutputSize());

        this->fc2->forward(layerOutputPtr, &layerOutputPtr);
        sigmoid(layerOutputPtr, this->fc2->getOutputSize());

        this->fc3->forward(layerOutputPtr, &layerOutputPtr);
        sigmoid(layerOutputPtr, this->fc3->getOutputSize());

        // Copy the final output from device to host memory
        hipMemcpy(output, layerOutputPtr, sizeof(float), hipMemcpyDeviceToHost);
    }

private:
    std::unique_ptr<Linear> fc1, fc2, fc3;
};

int main()
{
    // Creating cublas handler
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Here we will read weights from a given file
    FILE* fin = std::fopen("../task6/Weights/input", "rb");
    if (!fin)
    {
        std::exit(-1);
    }

    // Allocating memory for input data on host and device
    float* input, *devInput;
    hipHostMalloc(&input, sizeof(float) * 32 * 32);
    std::fread(input, sizeof(float), 32 * 32, fin);
    
    hipMalloc(&devInput, sizeof(float) * 32 * 32);
    hipMemcpy(devInput, input, sizeof(float) * 32 * 32, hipMemcpyHostToDevice);

    // Setting parameters for the network
    std::vector<LinearArguments> parameters = 
    {
        LinearArguments("../task6/Weights/weights1", 32 * 32, 16 * 16), 
        LinearArguments("../task6/Weights/weights2", 16 * 16, 4 * 4), 
        LinearArguments("../task6/Weights/weights3", 4 * 4, 1) 
    };

    // Creating an instance of our "network"
    Network* net =  new Network(handle, parameters);

    // Forward pass
    float out = 0.0;
    net->forward(devInput, &out);

    // See the result
    std::cout << "Output: " << out << std::endl;

    delete net;
    hipHostFree(input);
    hipFree(devInput);

    return 0;
}