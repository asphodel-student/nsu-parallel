
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void childKernel()
{
    printf("Hello ");
}

__global__ void tailKernel()
{
    printf("World!\n");
}

__global__ void parentKernel()
{
    // launch child
    childKernel<<<1,1>>>();
    if (hipSuccess != hipGetLastError()) {
        return;
    }

    // launch tail into cudaStreamTailLaunch stream
    // implicitly synchronizes: waits for child to complete
    tailKernel<<<1,1,0,cudaStreamTailLaunch>>>();

}

int main(int argc, char *argv[])
{
    // launch parent
    parentKernel<<<1,1>>>();
    if (hipSuccess != hipGetLastError()) {
        return 1;
    }

    // wait for parent to complete
    if (hipSuccess != hipDeviceSynchronize()) {
        return 2;
    }

    return 0;
}
