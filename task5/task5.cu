#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include <ctime>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include <mpi.h>

#define CORNER1 10
#define CORNER2 20
#define CORNER3 30
#define CORNER4 20

#define GET_CUDA_STATUS(status) { gpuAssert((status), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t status, const char *file, int line)
{
   if (status != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(status), file, line);
      std::exit(status);
   }
}

#define GET_MPI_STATUS(status) { mpiAssert((status), __FILE__, __LINE__); }
inline void mpiAssert(int status, const char *file, int line)
{
	if (status != MPI_SUCCESS)
	{
		fprintf(stderr, "MPIassert: %s %s %d\n", status, file, line);
		std::exit(status);
	}
}

// Заводим глобальные указатели для матриц
double 	*matrixA 			= nullptr, 
		*matrixB			= nullptr,
	 	*deviceMatrixAPtr 	= nullptr, 
		*deviceMatrixBPtr	= nullptr, 
		*deviceError 		= nullptr, 
		*errorMatrix 		= nullptr, 
		*tempStorage 		= nullptr;

void freeMemoryHandler()
{
	if (deviceMatrixAPtr) 	hipFree(deviceMatrixAPtr);
	if (deviceMatrixBPtr) 	hipFree(deviceMatrixBPtr);
	if (errorMatrix)	  	hipFree(errorMatrix);
	if (tempStorage) 		hipFree(tempStorage);
	if (matrixA) 			hipFree(matrixA);
	if (matrixB) 			hipFree(matrixB);
}

#define CALCULATE(matrixA, matrixB, size, i, j) \
	matrixB[i * size + j] = 0.25 * (matrixA[i * size + j - 1] + matrixA[(i - 1) * size + j] + \
			matrixA[(i + 1) * size + j] + matrixA[i * size + j + 1]);	

__global__
void calculateBoundaries(double* matrixA, double* matrixB, size_t size, size_t sizePerGpu)
{
	unsigned int idxUp = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idxDown = blockIdx.x * blockDim.x + threadIdx.x;

	if (idxUp == 0 || idxUp > size - 2) return;
	
	if(idxUp < size)
	{
		CALCULATE(matrixA, matrixB, size, 1, idxUp);
		CALCULATE(matrixA, matrixB, size, (sizePerGpu - 2), idxDown);
	}
}

// Главная функция - расчёт поля 
__global__
void calculateMatrix(double* matrixA, double* matrixB, size_t size, size_t sizePerGpu)
{
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(!(j < 1 || i < 2 || j > size - 2 || i > sizePerGpu - 2))
	{
		CALCULATE(matrixA, matrixB, size, i, j);
	}
}

// Функция, подсчитывающая разницу матриц
__global__
void getErrorMatrix(double* matrixA, double* matrixB, double* outputMatrix, size_t size, size_t sizePerGpu)
{
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

	size_t idx = i * size + j;
	if(!(j == 0 || i == 0 || j == size - 1 || i == sizePerGpu - 1))
	{
		outputMatrix[idx] = std::abs(matrixB[idx] - matrixA[idx]);
	}
}

int findNearestPowerOfTwo(size_t num) {
    int power = 1;
    while (power < num) {
        power <<= 1;
    }
    return power;
}

int main(int argc, char** argv)
{
	auto atExifStatus = std::atexit(freeMemoryHandler);
	if (atExifStatus != 0)
	{
		std::cout << "Register error" << std::endl;
		exit(-1);
	}

	if (argc != 4)
	{
		std::cout << "Invalid parameters" << std::endl;
		std::exit(-1);
	}

	// Получаем значения из командной строки
	const double minError = std::pow(10, -std::stoi(argv[1]));
	const int size = std::stoi(argv[2]);
	const int maxIter = std::stoi(argv[3]);
	const size_t totalSize = size * size;

	int rank, sizeOfTheGroup;
    GET_MPI_STATUS(MPI_Init(&argc, &argv));
    GET_MPI_STATUS(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    GET_MPI_STATUS(MPI_Comm_size(MPI_COMM_WORLD, &sizeOfTheGroup));

	int numOfDevices = 0;
	hipGetDeviceCount(&numOfDevices);
	if (sizeOfTheGroup > numOfDevices || sizeOfTheGroup < 1)
	{
		std::cout << "Invalid number of devices!";
		std::exit(-1);
	}

	GET_CUDA_STATUS(hipSetDevice(rank));

	if (rank == 0)
	{
		std::cout << "Parameters: " << std::endl <<
		"Min error: " << minError << std::endl <<
		"Maximal number of iteration: " << maxIter << std::endl <<
		"Grid size: " << size << std::endl;
	}

	// Размечаем границы между устройствами
	size_t sizeOfAreaForOneProcess = size / sizeOfTheGroup;
	size_t startYIdx = sizeOfAreaForOneProcess * rank;

	// Выделение памяти на хосте
    GET_CUDA_STATUS(hipHostMalloc(&matrixA, sizeof(double) * totalSize));
    GET_CUDA_STATUS(hipHostMalloc(&matrixB, sizeof(double) * totalSize));

	std::memset(matrixA, 0, size * size * sizeof(double));

	// Заполнение граничных условий
	matrixA[0] = CORNER1;
	matrixA[size - 1] = CORNER2;
	matrixA[size * size - 1] = CORNER3;
	matrixA[size * (size - 1)] = CORNER4;

	const double step = 1.0 * (CORNER2 - CORNER1) / (size - 1);
	for (int i = 1; i < size - 1; i++)
	{
		matrixA[i] = CORNER1 + i * step;
		matrixA[i * size] = CORNER1 + i * step;
		matrixA[size - 1 + i * size] = CORNER2 + i * step;
		matrixA[size * (size - 1) + i] = CORNER4 + i * step;
	}

	std::memcpy(matrixB, matrixA, totalSize * sizeof(double));

	// Расчитываем, сколько памяти требуется процессу
	if (rank != 0 && rank != sizeOfTheGroup - 1)
	{
		sizeOfAreaForOneProcess += 2;
	}
	else 
	{
		sizeOfAreaForOneProcess += 1;
	}

	size_t sizeOfAllocatedMemory = size * sizeOfAreaForOneProcess;

	// Выделяем память на девайсе
	GET_CUDA_STATUS(hipMalloc((void**)&deviceMatrixAPtr, sizeOfAllocatedMemory * sizeof(double)));
	GET_CUDA_STATUS(hipMalloc((void**)&deviceMatrixBPtr, sizeOfAllocatedMemory * sizeof(double)));
	GET_CUDA_STATUS(hipMalloc((void**)&errorMatrix, sizeOfAllocatedMemory * sizeof(double)));
	GET_CUDA_STATUS(hipMalloc((void**)&deviceError, sizeof(double)));

	// Копируем часть заполненной матрицы в выделенную память, начиная с 1 строки
	size_t offset = (rank != 0) ? size : 0;
 	GET_CUDA_STATUS(hipMemcpy(deviceMatrixAPtr, matrixA + (startYIdx * size) - offset, 
					sizeof(double) * sizeOfAllocatedMemory, hipMemcpyHostToDevice));
	GET_CUDA_STATUS(hipMemcpy(deviceMatrixBPtr, matrixB + (startYIdx * size) - offset, 
					sizeof(double) * sizeOfAllocatedMemory, hipMemcpyHostToDevice));

	// Здесь мы получаем размер временного буфера для редукции и выделяем память для этого буфера
	size_t tempStorageSize = 0;
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, size * sizeOfAreaForOneProcess);
	GET_CUDA_STATUS(hipMalloc((void**)&tempStorage, tempStorageSize));

	double* error;
	hipHostMalloc(&error, sizeof(double));
	*error = 1.0;

	hipStream_t stream, matrixCalculationStream;
	GET_CUDA_STATUS(hipStreamCreate(&stream));
	GET_CUDA_STATUS(hipStreamCreate(&matrixCalculationStream));

	unsigned int threads_x = std::min(findNearestPowerOfTwo(size), 1024);
    unsigned int blocks_y = sizeOfAreaForOneProcess;
    unsigned int blocks_x = size / threads_x;

    dim3 blockDim(threads_x, 1);
    dim3 gridDim(blocks_x, blocks_y);


	int iter = 0; 
	
	// Главный алгоритм 
	clock_t begin = clock();
	while((iter < maxIter) && (*error) > minError)
	{
		iter++;

		// Расчитываем границы, которые потом будем отправлять другим процессам
		calculateBoundaries<<<size, 1, 0, stream>>>(deviceMatrixAPtr, deviceMatrixBPtr, 
										size, sizeOfAreaForOneProcess);

		hipStreamSynchronize(stream);
		// Расчет матрицы
		calculateMatrix<<<gridDim, blockDim, 0, matrixCalculationStream>>>
							(deviceMatrixAPtr, deviceMatrixBPtr, size, sizeOfAreaForOneProcess);


		// Расчитываем ошибку каждую сотую итерацию
		if (iter % 100 == 0)
		{
			getErrorMatrix<<<gridDim, blockDim, 0, matrixCalculationStream>>>(deviceMatrixAPtr, deviceMatrixBPtr, errorMatrix,
															size, sizeOfAreaForOneProcess);

			hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, sizeOfAllocatedMemory, matrixCalculationStream);
			
			GET_CUDA_STATUS(hipStreamSynchronize(matrixCalculationStream));
			
			// Находим максимальную ошибку среди всех и передаём её всем процессам
			GET_MPI_STATUS(MPI_Allreduce((void*)deviceError, (void*)deviceError, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD));

			GET_CUDA_STATUS(hipMemcpyAsync(error, deviceError, sizeof(double), hipMemcpyDeviceToHost, matrixCalculationStream));
		}
		
		// Обмен "граничными" условиями каждой области
		// Обмен верхней границей
		if (rank != 0)
		{
		    GET_MPI_STATUS(MPI_Sendrecv(deviceMatrixBPtr + size + 1, size - 2, MPI_DOUBLE, rank - 1, 0, 
			deviceMatrixBPtr + 1, size - 2, MPI_DOUBLE, rank - 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE));
		}
		// Обмен нижней границей
		if (rank != sizeOfTheGroup - 1)
		{
		    GET_MPI_STATUS(MPI_Sendrecv(deviceMatrixBPtr + (sizeOfAreaForOneProcess - 2) * size + 1, size - 2, MPI_DOUBLE, rank + 1, 0,
							deviceMatrixBPtr + (sizeOfAreaForOneProcess - 1) * size + 1, 
							size - 2, MPI_DOUBLE, rank + 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE));
		}
	
		GET_CUDA_STATUS(hipStreamSynchronize(matrixCalculationStream));
		// Обмен указателей
		std::swap(deviceMatrixAPtr, deviceMatrixBPtr);
	}

	clock_t end = clock();
	if (rank == 0)
	{
		std::cout << "Time: " << 1.0 * (end - begin) / CLOCKS_PER_SEC << std::endl;
		std::cout << "Iter: " << iter << " Error: " << *error << std::endl;
	}

	GET_MPI_STATUS(MPI_Finalize());

	return 0;
}
