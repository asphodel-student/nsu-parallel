#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define CORNER1 10
#define CORNER2 20
#define CORNER3 30
#define CORNER4 20


// Главная функция - расчёт поля 
__global__
void calculateMatrix(double* matrixA, double* matrixB, size_t size)
{
	size_t i = blockIdx.x;
	size_t j = threadIdx.x;
	
	if(!(blockIdx.x == 0 || threadIdx.x == 0))
	{
		matrixB[i * size + j] = 0.25 * (matrixA[i * size + j - 1] + matrixA[(i - 1) * size + j] +
							matrixA[(i + 1) * size + j] + matrixA[i * size + j + 1]);		
	}
}

// Функция, подсчитывающая разницу матриц
__global__
void getErrorMatrix(double* matrixA, double* matrixB, double* outputMatrix)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(!(blockIdx.x == 0 || threadIdx.x == 0))
	{
		outputMatrix[idx] = std::abs(matrixB[idx] - matrixA[idx]);
	}
}


int main(int argc, char** argv)
{
	// Получаем значения из коммандной строки
	const double minError = std::pow(10, -std::stoi(argv[1]));
	const int size = std::stoi(argv[2]);
	const int maxIter = std::stoi(argv[3]);
	const size_t totalSize = size * size;

	std::cout << "Parameters: " << std::endl <<
		"Min error: " << minError << std::endl <<
		"Maximal number of iteration: " << maxIter << std::endl <<
		"Grid size: " << size << std::endl;

	// Выделение памяти на хосте
	double* matrixA = new double[totalSize];
	double* matrixB = new double[totalSize];
	
	std::memset(matrixA, 0, totalSize * sizeof(double));

	// Заполнение граничных условий
	matrixA[0] = CORNER1;
	matrixA[size - 1] = CORNER2;
	matrixA[size * size - 1] = CORNER3;
	matrixA[size * (size - 1)] = CORNER4;

	const double step = 1.0 * (CORNER2 - CORNER1) / (size - 1);
	for (int i = 1; i < size - 1; i++)
	{
		matrixA[i] = CORNER1 + i * step;
		matrixA[i * size] = CORNER1 + i * step;
		matrixA[size - 1 + i * size] = CORNER2 + i * step;
		matrixA[size * (size - 1) + i] = CORNER4 + i * step;
	}

	std::memcpy(matrixB, matrixA, totalSize * sizeof(double));

	// Выбор устройства
	hipSetDevice(3);

	// Выделяем папять на девайсе и копируем память с хоста
	double* deviceMatrixAPtr, *deviceMatrixBPtr, *deviceError, *errorMatrix, *tempStorage = NULL;
	size_t tempStorageSize = 0;

	hipError_t cudaStatus_1 = hipMalloc((void**)(&deviceMatrixAPtr), sizeof(double) * totalSize);
	hipError_t cudaStatus_2 = hipMalloc((void**)(&deviceMatrixBPtr), sizeof(double) * totalSize);
	hipMalloc((void**)&deviceError, sizeof(double));
	cudaStatus_1 = hipMalloc((void**)&errorMatrix, sizeof(double) * totalSize);
	
	if (cudaStatus_1 != 0 || cudaStatus_2 != 0)
	{
		std::cout << "Memory allocation error" << std::endl;
		return -1;
	}

	cudaStatus_1 = hipMemcpy(deviceMatrixAPtr, matrixA, sizeof(double) * totalSize, hipMemcpyHostToDevice);
	cudaStatus_2 = hipMemcpy(deviceMatrixBPtr, matrixB, sizeof(double) * totalSize, hipMemcpyHostToDevice);

	if (cudaStatus_1 != 0 || cudaStatus_2 != 0)
	{
		std::cout << "Memory transfering error" << std::endl;
		return -1;
	}

	// Здесь мы получаем размер временного буфера для редукции
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, totalSize);
	
	// Выделяем память для буфера
	hipMalloc((void**)&tempStorage, tempStorageSize);

	int iter = 0; 
	double error = 1.0;

	// Главный алгоритм 
	clock_t begin = clock();
	while(iter < maxIter && error > minError)
	{
		iter++;
		// Расчет матрицы
		calculateMatrix<<<size - 1, size - 1>>>(deviceMatrixAPtr, deviceMatrixBPtr, size);

		// Расчитываем ошибку каждую сотую итерацию
		if(iter % 100 == 0)
		{
			getErrorMatrix<<<size - 1, size - 1>>>(deviceMatrixAPtr, deviceMatrixBPtr, errorMatrix);
			hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, totalSize);
			hipMemcpy(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost);
		}
		
		// Обмен указателей
		std::swap(deviceMatrixAPtr, deviceMatrixBPtr);
	}

	clock_t end = clock();
	std::cout << "Time: " << 1.0 * (end - begin) / CLOCKS_PER_SEC << std::endl;
	std::cout << "Iter: " << iter << " Error: " << error << std::endl;

	// Высвобождение памяти
	hipFree(deviceMatrixAPtr);
	hipFree(deviceMatrixBPtr);
	hipFree(errorMatrix);
	hipFree(tempStorage);

	delete[] matrixA;
	delete[] matrixB;

	return 0;
}
